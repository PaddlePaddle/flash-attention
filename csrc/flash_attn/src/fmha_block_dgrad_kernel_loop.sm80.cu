#include "hip/hip_runtime.h"
/* Copyright (c) 2022, Tri Dao.
 */
#include "fmha.h"
#include "static_switch.h"
#include "fmha_block_dgrad_kernel_1xN_loop.h"
#include "cuda_utils.h"

template<typename Kernel_traits>
__global__ void fmha_bwd_dot_do_o_kernel(FMHA_dgrad_params params) {
    fmha::compute_dot_do_o<Kernel_traits>(params);
}

template<typename Kernel_traits, bool Is_dropout, bool Is_causal, int loop_steps=-1>
__global__ void fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel(FMHA_dgrad_params params) {
    fmha::compute_block_dq_dk_dv_1xN<Kernel_traits, Is_dropout, Is_causal, loop_steps>(params);
}
template<typename Kernel_traits, bool Is_dropout, bool Is_causal>
__global__ void fmha_block_dgrad_sm80_dq_dk_dv_loop_seqparallel_kernel(FMHA_dgrad_params params) {
    fmha::compute_block_dq_dk_dv_1xN_seqparallel<Kernel_traits, Is_dropout, Is_causal>(params);
}
inline int num_splits_heuristic_bwd(int batch_nheads, int num_SMs, int ctas_per_sm, int seqlen,
        int blocksize, bool is_causal) {
    float n_waves_1 = float(batch_nheads) / (num_SMs * ctas_per_sm);
    float eff_1 = n_waves_1 / ceil(n_waves_1);
    int num_splits_parallel = seqlen / blocksize;
    float n_waves_parallel = float(batch_nheads * num_splits_parallel) / (num_SMs * ctas_per_sm);
    float eff_parallel_raw = n_waves_parallel / ceil(n_waves_parallel);
    float discount_factor;
    if (!is_causal) {
        discount_factor = 1.f + float(blocksize) / seqlen;
    } else {  // For causal, parallelizing seems to help with load-balancing as well
        // For example, if headdim=128, seqlen >= 1280 always prefers parallel
        if (seqlen / blocksize >= 10) return num_splits_parallel;
        discount_factor = 1.f + 0.5 * float(blocksize) / seqlen;
    }
    float eff_parallel = eff_parallel_raw / discount_factor;
    return eff_1 >= eff_parallel ? 1 : num_splits_parallel;
}

template<typename Kernel_traits>
void run_fmha_block_dgrad_sm80_loop_(FMHA_dgrad_params &params, hipStream_t stream) {
    constexpr int smem_size_softmax = Kernel_traits::Cta_tile_p::M * Kernel_traits::Cta_tile_p::WARPS_N * sizeof(float);
    constexpr int smem_size_q = Kernel_traits::Smem_tile_q::BYTES_PER_TILE;
    constexpr int smem_size_v = Kernel_traits::Smem_tile_v::BYTES_PER_TILE;
    constexpr int smem_size_dq = Kernel_traits::Smem_tile_o::BYTES_PER_TILE;
    constexpr int smem_size_dp_sum = Kernel_traits::Smem_dp_sum::BYTES_PER_TILE;

    using Smem_tile_s = fmha::Smem_tile_mma_transposed<typename Kernel_traits::Cta_tile_p>;
    constexpr int smem_size_s = Smem_tile_s::BYTES_PER_TILE;
    static_assert(smem_size_s == 16 * Kernel_traits::Cta_tile_p::N * 2);
    static_assert(smem_size_dq == 16 * Kernel_traits::Cta_tile_p::K * 4 * Kernel_traits::Cta_tile_p::WARPS_N);
    static_assert(smem_size_dp_sum == 16 * 4 * 2);

    constexpr int smem_size_dq_dk_dv = smem_size_q * 2 + smem_size_v * (Kernel_traits::V_IN_REGS ? 1 : 2) + smem_size_dq + smem_size_s * 2 + smem_size_dp_sum;

    bool is_dropout = params.p_dropout < 1.f;  // params.p_dropout is the probability of "keeping"
    bool is_causal = params.is_causal;
    BOOL_SWITCH(is_dropout, IsDropoutConst, ([&] {
        auto kernel = is_dropout
            ? (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, true> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, false>)
            : (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, true> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, false>);
        constexpr int blocksize_c = Kernel_traits::Cta_tile_p::N;
        if (params.seqlen_k == blocksize_c) {
            kernel = is_dropout
                ? (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, true, /*loop_steps=*/1> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, false, /*loop_steps=*/1>)
                : (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, true, /*loop_steps=*/1> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, false, /*loop_steps=*/1>);
        } else if (params.seqlen_k == blocksize_c * 2) {
            kernel = is_dropout
            ? (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, true, /*loop_steps=*/2> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, true, false, /*loop_steps=*/2>)
            : (is_causal ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, true, /*loop_steps=*/2> : &fmha_block_dgrad_sm80_dq_dk_dv_loop_kernel<Kernel_traits, false, false, /*loop_steps=*/2>);
        }

        auto kernel_seqparallel = params.is_causal
            ? &fmha_block_dgrad_sm80_dq_dk_dv_loop_seqparallel_kernel<Kernel_traits, IsDropoutConst, true>
            : &fmha_block_dgrad_sm80_dq_dk_dv_loop_seqparallel_kernel<Kernel_traits, IsDropoutConst, false>;

        if( smem_size_dq_dk_dv >= 48 * 1024 ) {
            FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size_dq_dk_dv));
            FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    kernel_seqparallel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size_dq_dk_dv));
        }
    //dim3 grid(params.b, params.h);
    //kernel<<<grid, Kernel_traits::THREADS, smem_size_dq_dk_dv, stream>>>(params);
    //FMHA_CHECK_CUDA(hipPeekAtLastError());
        // Automatically set num_splits to maximize occupancy
        if (params.num_splits <= 0) {
            int ctas_per_sm;
            hipError_t status_ = hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &ctas_per_sm, kernel, Kernel_traits::THREADS, smem_size_dq_dk_dv);
            auto dprops = GetDeviceProperties(-1);
            // printf("CTAS_PER_SM = %d, nSMs = %d\n", ctas_per_sm, dprops->multiProcessorCount);
            constexpr int M = Kernel_traits::Cta_tile_p::M;
            // We don't want more than 10 splits due to numerical error.
            // Numerical error on dk/dv scales as sqrt(num_splits).
            params.num_splits = num_splits_heuristic_bwd(
                params.b * params.h, dprops->multiProcessorCount,
                ctas_per_sm, params.seqlen_k, blocksize_c, params.is_causal
            );
        }
        //if (configure) return;
        if (params.num_splits == 1) {
            dim3 grid(params.b, params.h, params.num_splits);
            kernel<<<grid, Kernel_traits::THREADS, smem_size_dq_dk_dv, stream>>>(params);
        } else {
            dim3 grid_dot(params.b, params.h, (params.seqlen_q + 128 - 1) / 128);
            fmha_bwd_dot_do_o_kernel<Kernel_traits><<<grid_dot, Kernel_traits::THREADS, 0, stream>>>(params);
            int num_splits = params.seqlen_k / blocksize_c;  // seqlen_k is divisible by blocksize_c
            dim3 grid(params.b, params.h, num_splits);
            kernel_seqparallel<<<grid, Kernel_traits::THREADS, smem_size_dq_dk_dv, stream>>>(params);
        }
        FMHA_CHECK_CUDA(hipPeekAtLastError());
    }));
}

void run_fmha_block_dgrad_sm80(FMHA_dgrad_params &params, hipStream_t stream) {
    FP16_SWITCH(params.is_bf16, ([&] {
        if (params.d == 16) {
            using Kernel_traits = FMHA_kernel_traits<256, 16, 16, 1, 8, 0x08u, elem_type>;
            run_fmha_block_dgrad_sm80_loop_<Kernel_traits>(params, stream);
        } else if (params.d == 32) {
            using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 8, 0x08u, elem_type>;
            run_fmha_block_dgrad_sm80_loop_<Kernel_traits>(params, stream);
        } else if (params.d == 64) {
            using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 8, 0x100u, elem_type>;
            run_fmha_block_dgrad_sm80_loop_<Kernel_traits>(params, stream);
        } else if (params.d == 128) {
            using Kernel_traits = FMHA_kernel_traits<256, 128, 16, 1, 8, 0x08u, elem_type>;
            run_fmha_block_dgrad_sm80_loop_<Kernel_traits>(params, stream);
        }
    }));
}
